#include "hip/hip_runtime.h"
#include "cuda.cuh"

namespace SOM {
	
	void SOM::cudaCalc::activiteNeurone(double * d_out, double * d_in, double * d_w, const int ARRAYSIZE)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		float _act = 0;
		float f = d_in[idx];
		float f2 = d_w[idx];

		if (idx < ARRAYSIZE) {
			_act += pow(f - f2, 2);
			d_out[idx] = sqrt(_act);
		}
	}

	double SOM::cudaCalc::launchActiviteNeurone(const double* h_actIn) const
	{
		const int ARRAYSIZE = 10; //grid dimension
		const int ARRAY_BYTES = ARRAYSIZE * sizeof(double); //memoire � allouer par element

		// genere les tableaux sur le host pour le calcul d'activit�
		double h_weights[ARRAYSIZE], h_actOUT[ARRAYSIZE];

		// d�claration des pointers m�moire sur GPU
		float * d_actIn;
		float * d_weights;
		float * d_actOut;

		// alloue la memoire sur GPU pour les pointers
		hipMalloc((void**)&d_actIn, ARRAY_BYTES);
		hipMalloc((void**)&d_weights, ARRAY_BYTES);
		hipMalloc((void**)&d_actOut, ARRAY_BYTES);

		// transfert les tableaux sur le GPU
		hipMemcpy(d_actIn, h_actIn, ARRAY_BYTES, hipMemcpyHostToDevice);
		hipMemcpy(d_weights, h_weights, ARRAY_BYTES, hipMemcpyHostToDevice);

		// lance le kernel activiteNeurone
		activiteNeurone << <ARRAYSIZE, 3 >> > (d_actOut, d_actIn, d_weights);

		// copie le r�sultat final sur le CPU
		hipMemcpy(h_actOUT, d_actOut, ARRAY_BYTES, hipMemcpyDeviceToHost);

		// affiche le tableau de poids final
		for (int i = 0; i < ARRAYSIZE; i++) {
			printf("weight:%f", h_actOUT[i]);
			printf(((i % 4) != 3) ? "\t" : "\n");
		}

		hipFree(d_actIn);
		hipFree(d_weights);
		hipFree(d_actOut);
		_getch();

		return *h_actOUT;
	}

}