#include "hip/hip_runtime.h"
#include "cuda.cuh"

namespace SOM {
	__global__
	void activiteNeurone(double * d_out, double * d_in, double * d_w, const int ARRAYSIZE)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		float _act = 0;
		float f = d_in[idx];
		float f2 = d_w[idx];

		if (idx < ARRAYSIZE) {
			_act += pow(f - f2, 2);
			d_out[idx] = sqrt(_act);
		}
	}
	
	double launchActiviteNeurone(double* h_actIn)
	{
		const int ARRAYSIZE = 10; //grid dimension
		const int ARRAY_BYTES = ARRAYSIZE * sizeof(double); //memoire � allouer par element

		// tableaux sur le host pour le calcul d'activit�
		double h_weights[ARRAYSIZE], h_actOUT[ARRAYSIZE];

		// d�claration des pointers m�moire sur GPU
		double * d_actIn;
		double * d_weights;
		double * d_actOut;

		// alloue la memoire sur GPU pour les pointers
		hipMalloc((void**)&d_actIn, ARRAY_BYTES);
		hipMalloc((void**)&d_weights, ARRAY_BYTES);
		hipMalloc((void**)&d_actOut, ARRAY_BYTES);

		// transfert les tableaux sur le GPU
		hipMemcpy(d_actIn, h_actIn, ARRAY_BYTES, hipMemcpyHostToDevice);
		hipMemcpy(d_weights, h_weights, ARRAY_BYTES, hipMemcpyHostToDevice);

		// lance le kernel activiteNeurone
		activiteNeurone <<<ARRAYSIZE, 3 >>> (d_actOut, d_actIn, d_weights, ARRAYSIZE);

		// copie le r�sultat final sur le CPU
		hipMemcpy(h_actOUT, d_actOut, ARRAY_BYTES, hipMemcpyDeviceToHost);

		/*
		// affiche le tableau de poids final
		for (int i = 0; i < ARRAYSIZE; i++) {
			printf("weight:%f", h_actOUT[i]);
			printf(((i % 4) != 3) ? "\t" : "\n");
		}*/

		hipFree(d_actIn);
		hipFree(d_weights);
		hipFree(d_actOut);
		_getch();

		return *h_actOUT;
	}

}