#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <conio.h>
#include <math.h>

#define ARRAYSIZE 10

__global__ void vectDist(float * d_out, float * d_in, float * d_in2) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	float _res = 0;
	float f = d_in[idx];
	float f2 = d_in2[idx];
	if (idx < ARRAYSIZE) {
		_res += sqrt(pow(f - f2, 2));
		d_out[idx] = _res;
	}
}

int main(int argc, char ** argv) {
	const int ARRAY_BYTES = ARRAYSIZE * sizeof(float);

	// generate the input array on the host
	float h_in[ARRAYSIZE], h_weights[ARRAYSIZE], h_distOUT[ARRAYSIZE];

	for (int i = 0; i < ARRAYSIZE; i++) {
		h_in[i] = float(i+1);
		h_weights[i] = float(i);		
	}
	//sqrt(pow(x-x, 2)+pow(y-y,2)   distance formula

	/*for (int i = 0; i < ARRAYSIZE; ++i) {
		h_distOUT[i] = sqrt(pow(h_in[i] - h_weights[i], 2));
	}*/


	// declare GPU memory pointers
	float * d_in;
	float * d_in2;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**)&d_in, ARRAY_BYTES);
	hipMalloc((void**)&d_in2, ARRAY_BYTES);  
	hipMalloc((void**)&d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_in2, h_weights, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	vectDist <<<ARRAYSIZE, 3>> >(d_out, d_in, d_in2);

	// copy back the result array to the CPU
	hipMemcpy(h_distOUT, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print out the resulting array
	for (int i = 0; i < ARRAYSIZE; i++) {
		printf("weight:%f", h_distOUT[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}

	hipFree(d_in);
	hipFree(d_in2);
	hipFree(d_out);
	_getch();
	return 0;
}