
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <stdio.h>
//#include <random>
//#include <conio.h>
//
//
//#define N 2048
//#define NB_THREADS 1024
//
//__global__ void multVect(int* result, int* a, int* b) {
//	int idx = threadIdx.x + blockIdx.x * blockDim.x;
//	if(idx <N)	result[idx] = a[idx] * b[idx];
//}
//
//
//int main() {
//	
//	int *d_vect1, *d_vect2, *d_vect3;
//
//	const int size = N * sizeof(int);
//
//	cudaMallocManaged(&d_vect1, size);
//	cudaMallocManaged(&d_vect2, size);
//	cudaMallocManaged(&d_vect3, size);
//
//	for (int i = 0; i < N; ++i)
//	{
//		d_vect1[i] = rand() % (int)floor(sqrt(INT_MAX));
//		d_vect2[i] = rand() % (int)floor(sqrt(INT_MAX));
//	}
//
//
//	multVect <<<N/ NB_THREADS, NB_THREADS >> >(d_vect3, d_vect1, d_vect2);
//
//	cudaDeviceSynchronize();
//
//	for (int i = 0; i < N; ++i)
//	{
//		printf("%d*%d=%d ", d_vect1[i], d_vect2[i], d_vect3[i]);
//	}
//
//	cudaFree(d_vect1);
//	cudaFree(d_vect2);
//	cudaFree(d_vect3);
//
//	_getch();
//	return 0;
//}