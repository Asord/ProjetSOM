
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <stdio.h>
//#include <random>
//#include <conio.h>
//
//
//#define N 2048
//#define NB_THREADS 1024
//
//__global__ void multVect(int* result, int* a, int* b) {
//	int idx = threadIdx.x + blockIdx.x * blockDim.x;
//	if(idx < N)
//	result[idx] = a[idx] * b[idx];
//}
//
//
//int main() {
//	int *h_vect1, *h_vect2, *h_vect3;
//	int *d_vect1, *d_vect2, *d_vect3;
//
//	const int size = N * sizeof(int);
//
//	cudaMalloc(&d_vect1, size);
//	cudaMalloc(&d_vect2, size);
//	cudaMalloc(&d_vect3, size);
//
//	h_vect1 = (int *)malloc(size); 
//	h_vect2 = (int *)malloc(size); 
//	h_vect3 = (int *)malloc(size);
//
//	for(int i = 0; i < N; ++i)
//	{
//		h_vect1[i]= rand() % (int)floor(sqrt(INT_MAX));
//		h_vect2[i]= rand() % (int)floor(sqrt(INT_MAX));
//	}
//	
//	cudaMemcpy(d_vect1, h_vect1, size, cudaMemcpyHostToDevice);
//	cudaMemcpy(d_vect2, h_vect2, size, cudaMemcpyHostToDevice);
//
//	multVect <<<N / NB_THREADS, NB_THREADS >> >(d_vect3, d_vect1, d_vect2);
//
//	cudaMemcpy(h_vect3, d_vect3, size, cudaMemcpyDeviceToHost);
//
//	for (int i = 0; i < N; ++i) 
//	{
//		printf("%d*%d=%d", h_vect1[i], h_vect2[i], h_vect3[i]);
//	}
//
//	free(h_vect1);
//	free(h_vect2);
//	free(h_vect3);
//    cudaFree(d_vect1);
//    cudaFree(d_vect2);
//    cudaFree(d_vect3);
//	
//	_getch();
//	return 0;
//}